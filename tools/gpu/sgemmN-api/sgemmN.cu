
#include <hip/hip_runtime.h>
__device__ void saxpy( float a, float *b, float *c )
{
	c[0] += a*b[0];
	c[1] += a*b[1];
	c[2] += a*b[2];
	c[3] += a*b[3];
	c[4] += a*b[4];
	c[5] += a*b[5];
	c[6] += a*b[6];
	c[7] += a*b[7];
	c[8] += a*b[8];
	c[9] += a*b[9];
	c[10] += a*b[10];
	c[11] += a*b[11];
	c[12] += a*b[12];
	c[13] += a*b[13];
	c[14] += a*b[14];
	c[15] += a*b[15];
}

extern "C" __global__ void sgemmNT( const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta )
{
	const int inx = threadIdx.x;
	const int iny = threadIdx.y;
	const int ibx = blockIdx.x * 64;
	const int iby = blockIdx.y * 16;
	const int id  = inx + iny*16;

	A += ibx + id;
	B += iby + inx + __mul24( iny, ldb );
	C += ibx + id  + __mul24( iby, ldc );
	
	float a[4] = {A[0], A[lda], A[2*lda], A[3*lda]};
	float b = B[0];
	
	const float *Blast = B + k*ldb;

	A += 4*lda;
	B += 4*ldb;
    
	__shared__ float bs[4][16];
	float c[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    
	do
	{
		float as[4] = {a[0], a[1], a[2], a[3]};
		
		bs[iny][inx] = b;
		__syncthreads();
		
		a[0] = A[0*lda];
		a[1] = A[1*lda];
		a[2] = A[2*lda];
		a[3] = A[3*lda];
		b    = B[0];
		
		saxpy( as[0], &bs[0][0], c );
		saxpy( as[1], &bs[1][0], c );
		saxpy( as[2], &bs[2][0], c );
		saxpy( as[3], &bs[3][0], c );
		
		A += 4*lda;
		B += 4*ldb;
		__syncthreads();
		
	} while( B < Blast );
	
	bs[iny][inx] = b;
	__syncthreads();
	
	saxpy( a[0], &bs[0][0], c );
	saxpy( a[1], &bs[1][0], c );
	saxpy( a[2], &bs[2][0], c );
	saxpy( a[3], &bs[3][0], c );

	for( int i = 0; i < 16; i++, C += ldc )
		C[0] = alpha*c[i] + beta*C[0];
}	

extern "C" __global__ void sgemmNN( const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta )
{
	const int inx = threadIdx.x;
	const int iny = threadIdx.y;
	const int ibx = blockIdx.x * 64;
	const int iby = blockIdx.y * 16;
	const int id = inx + iny*16;
	
	A += ibx + id;
	B += inx + __mul24( iby + iny, ldb );
	C += ibx + id  + __mul24( iby, ldc );
	
	const float *Blast = B + k;
	
	float c[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    
	do
	{
		float a[4] = { A[0*lda], A[1*lda], A[2*lda], A[3*lda] };

		__shared__ float bs[16][17];
		bs[inx][iny]    = B[0*ldb];
		bs[inx][iny+4]  = B[4*ldb];
		bs[inx][iny+8]  = B[8*ldb];
		bs[inx][iny+12] = B[12*ldb];
		__syncthreads();

		A += 4*lda;
		saxpy( a[0], &bs[0][0], c );		a[0] = A[0*lda];
		saxpy( a[1], &bs[1][0], c );		a[1] = A[1*lda];
		saxpy( a[2], &bs[2][0], c );		a[2] = A[2*lda];
		saxpy( a[3], &bs[3][0], c );		a[3] = A[3*lda];

		A += 4*lda;
		saxpy( a[0], &bs[4][0], c );		a[0] = A[0*lda];
		saxpy( a[1], &bs[5][0], c );		a[1] = A[1*lda];
		saxpy( a[2], &bs[6][0], c );		a[2] = A[2*lda];
		saxpy( a[3], &bs[7][0], c );		a[3] = A[3*lda];

		A += 4*lda;
		saxpy( a[0], &bs[8][0], c );		a[0] = A[0*lda];
		saxpy( a[1], &bs[9][0], c );		a[1] = A[1*lda];
		saxpy( a[2], &bs[10][0], c );		a[2] = A[2*lda];
		saxpy( a[3], &bs[11][0], c );		a[3] = A[3*lda];

		A += 4*lda;
		saxpy( a[0], &bs[12][0], c );
		saxpy( a[1], &bs[13][0], c );
		saxpy( a[2], &bs[14][0], c );
		saxpy( a[3], &bs[15][0], c );
		
		B += 16;
		__syncthreads();
	} while( B < Blast );
	
	for( int i = 0; i < 16; i++, C += ldc )
                C[0] = alpha*c[i] + beta*C[0];
}	
